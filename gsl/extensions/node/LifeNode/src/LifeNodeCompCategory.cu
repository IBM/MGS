#include "hip/hip_runtime.h"
#ifndef LIFENODECOMPCATEGORY_CU
#define LIFENODECOMPCATEGORY_CU

void __global__ LifeNode_kernel_initialize(
     int* value,
     int* publicValue,
   //#if DATAMEMBER_ARRAY_ALLOCATION == OPTION_3
   //  ShallowArray_Flat<int*, Array_Flat<int>::MemLocation::UNIFIED_MEM>* neighbors,  
   //#elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4
   //  int** neighbors,  
   //#elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4b
   //  int** neighbors,  
   //#elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_5
   //  int** neighbors,  
   //#endif
     //RNG& rng
     unsigned size,
     int tooSparse,
     int tooCrowded
     ) 
{
   int index =  blockDim.x * blockIdx.x + threadIdx.x;
   if (index < size)
   {
      publicValue[index]=value[index];
   }
}
void __global__ LifeNode_kernel_update(
     int* value,
     int* publicValue,
   #if DATAMEMBER_ARRAY_ALLOCATION == OPTION_3
     ShallowArray_Flat<int*, Array_Flat<int>::MemLocation::UNIFIED_MEM>* neighbors,  
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4
     int** neighbors,  
     int* neighbors_start_offset,  
     int* neighbors_num_elements,  
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4b
     int** neighbors,  
     int neighbors_max_elements,
     int* neighbors_num_elements,  
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_5
     int** neighbors,  
   #endif
     // //RNG& rng
      unsigned size
     , int tooSparse 
     , int tooCrowded
      ) 
{
   int index =  blockDim.x * blockIdx.x + threadIdx.x;
   if (index < size)
   {
      int neighborCount=0;
      /* TUAN TODO find out the bug in here */
   #if DATAMEMBER_ARRAY_ALLOCATION == OPTION_3
      ShallowArray_Flat<int*>::iterator end = neighbors[index].end();
      for (auto iter=neighbors[index].begin(); iter!=end; ++iter) {
         neighborCount += **iter;
      }
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4
      auto um_neighbors_from = neighbors_start_offset[index];
      auto um_neighbors_to = um_neighbors_num_elements[index]-1;
      for (auto idx = um_neighbors_from; idx < um_neighbors_to; ++idx) {
         neighborCount += *(neighbors[idx]);
      }
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4b
      auto um_neighbors_from = index * neighbors_max_elements;
      auto um_neighbors_to = um_neighbors_from + neighbors_num_elements[index];
      for (auto idx = um_neighbors_from; idx < um_neighbors_to; ++idx) {
         neighborCount += *(neighbors[idx]);
      }
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_5
   #endif

      if (neighborCount<= tooSparse || neighborCount>= tooCrowded) {
         value[index]=0;
      }
      else {
         value[index]=1;
      }
   }
}

void __global__ LifeNode_kernel_copy(
     int* value,
     int* publicValue,
     float* weight,
     float* publicWeight,
     unsigned size
     ) 
{
   int index =  blockDim.x * blockIdx.x + threadIdx.x;
   if (index < size)
   {
      publicValue[index]=value[index];
      publicWeight[index]=weight[index];
   }
}

void __global__ LifeNode_kernel_updateWeight(
   int* value
   , int* publicValue
   , float* weight
   , float* publicWeight
   #if DATAMEMBER_ARRAY_ALLOCATION == OPTION_3
   , ShallowArray_Flat<int*, Array_Flat<int>::MemLocation::UNIFIED_MEM>* neighbors
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4
   , int** neighbors
   , int* neighbors_start_offset
   , int* neighbors_num_elements
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4b
   , int** neighbors
   , int neighbors_max_elements
   , int* neighbors_num_elements
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_5
   , ShallowArray< int* >* neighbors
   //need more info here
   #endif

   #if DATAMEMBER_ARRAY_ALLOCATION == OPTION_3
   , ShallowArray_Flat<float*, Array_Flat<int>::MemLocation::UNIFIED_MEM>* neighborsWeight
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4
   , float** neighborsWeight
   , int* neighborsWeight_start_offset
   , int* neighborsWeight_num_elements
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_4b
   , float** neighborsWeight
   , int neighborsWeight_max_elements
   , int* neighborsWeight_num_elements
   #elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_5
   , ShallowArray< float* >* neighborsWeight
   //need more info here
   #endif

   , unsigned size
   , int complexity
   , int actionType
   , int tooCrowded
   , int tooSparse
)
{
   int index = blockDim.x * blockIdx.x + threadIdx.x;
   if (index < size) {
      float weightSum = 0;
      float learnRate = 0.0001;
      float dw = 0;
       // add your code here
      for (int ii = 0; ii < complexity; ii++)
      {
#if DATAMEMBER_ARRAY_ALLOCATION == OPTION_4b
         auto um_neighborsWeight_from = index * neighborsWeight_max_elements;
         auto um_neighborsWeight_to = um_neighborsWeight_from + neighborsWeight_num_elements[index];
         for (auto idx = um_neighborsWeight_from; idx < um_neighborsWeight_to; ++idx) {
            weightSum += *(neighborsWeight[idx]);
         }
#elif DATAMEMBER_ARRAY_ALLOCATION == OPTION_5
#endif
         if (actionType == F_SIGMOID)
            dw += sigmoid(weightSum);
         else if (actionType == F_ReLU)
            dw += ReLU(weightSum);
         else if (actionType == F_TANH)
            dw += tanh(weightSum);

         if (value[index] == 1) {
            weight[index] += dw;
         }
         else {
            weight[index] -= dw;
         }
      }
   }
}
#endif
